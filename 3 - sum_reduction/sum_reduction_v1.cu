
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cmath>

//TB_SIZE
#define SIZE 256

enum {FAIL, SUCCESS};

void init(int* vec,const int len)
{
  for(int i=0;i<len;i++) vec[i] = 1;
}

__global__ void sum_reduction(int* v, int* result)
{
  /*
    PROBLEMS
    We are not fully utilizing threads.
    Namely, there are warp divergence problem. (Threads with odd idx are idle!)
    Additionally, division operation is costly in GPU!

    We will gradually optimize this algorithm in this repo! ;)
  */
  __shared__ int partial_sum[SIZE];

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  partial_sum[threadIdx.x] = v[tid];
  __syncthreads();

  for(int stride=1; stride< blockDim.x; stride*= 2)
  {
    if(threadIdx.x % (2*stride) == 0)
    {
      partial_sum[threadIdx.x] += partial_sum[threadIdx.x + stride];
    }
    __syncthreads();
  }

  if(threadIdx.x == 0) result[blockIdx.x] = partial_sum[0];
}

int main(void)
{
  int n = 1<<16;
  size_t size = n * sizeof(int);

  int *h_v, *h_result;
  int *d_v, *d_result;

  h_v = new int[n];
  h_result = new int[n];
  hipMalloc(&d_v,size);
  hipMalloc(&d_result,size);

  //Initialize vector
  init(h_v, n);

  hipMemcpy(d_v, h_v, size, hipMemcpyHostToDevice);

  int TB_SIZE = SIZE;
  int GRID_SIZE = (int)ceil((float)n/TB_SIZE);

  sum_reduction <<<GRID_SIZE, TB_SIZE>>> (d_v, d_result);
  sum_reduction <<<1, GRID_SIZE>>> (d_result, d_result);

  hipMemcpy(h_result,d_result,size, hipMemcpyDeviceToHost);

  std::cout << "Accumulated result : " << h_result[0] << std::endl;
  
  delete[] h_v;
  delete[] h_result;
  hipFree(d_v);
  hipFree(d_result);

  return 0;
}