
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cmath>
#include <random>

enum {FAIL, SUCCESS};

int verifier(int *arr,int *mask, int *res,int n,int m)
{
  int rad = m/2;
  for(int i=0;i<n;i++)
  {
    int tmp = 0;
    int s = i-rad;
    for(int j=0;j<m;j++)
    {
      if(s+j >=0 && s+j < n) tmp += arr[s+j] * mask[j];
    }
    if(tmp != res[i]) return FAIL;
  }

  return SUCCESS;
}

void init_vector(int* vec,const int len)
{
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<int> dis(0, 99);
    
    for(int i=0;i<len;i++) vec[i] = dis(gen);
}

__global__ void convolution_1d(int* arr, int* mask, int* res, int n, int m)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if(tid>=n) return;

  int rad = m/2;
  int s = tid - rad;

  int tmp = 0;
  for(int i=0;i<m;i++)
  {
    if(s+i >=0 && s+i < n) tmp += arr[s+i] * mask[i];
  }
  res[tid] = tmp;
}

int main(void)
{
  int n = 1 << 20;
  int size_n = sizeof(int) * n;

  int m = 7;  //mask size
  int size_m = sizeof(int) * m;

  int *h_arr = new int[n];
  int *h_mask = new int[m];
  int *h_result = new int[n];

  init_vector(h_arr,n);
  init_vector(h_mask,m);

  int *d_array, *d_mask, *d_result;

  hipMalloc(&d_array,size_n);
  hipMalloc(&d_mask,size_m);
  hipMalloc(&d_result,size_n);

  hipMemcpy(d_array,h_arr,size_n,hipMemcpyHostToDevice);
  hipMemcpy(d_mask,h_mask,size_m,hipMemcpyHostToDevice);

  int THREADS = 256;
  int GRID = (int)ceil((float)n/THREADS);

  convolution_1d <<< GRID, THREADS>>>(d_array,d_mask,d_result,n,m);

  hipMemcpy(h_result,d_result,size_n,hipMemcpyDeviceToHost);
  int val = verifier(h_arr,h_mask,h_result,n,m);

  if(val == SUCCESS) std::cout << "YAY!!" << std::endl;
  else std::cout << "Hmm..." << std::endl;

  delete[] h_arr;
  delete[] h_mask;
  delete[] h_result;
  hipFree(d_array);
  hipFree(d_mask);
  hipFree(d_result);
  return 0;
}